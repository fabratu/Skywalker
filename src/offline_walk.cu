#include "hip/hip_runtime.h"
/*
 * @Description: just perform RW
 * @Date: 2020-11-30 14:30:06
 * @LastEditors: Pengyu Wang
 * @LastEditTime: 2022-03-03 22:46:47
 * @FilePath: /skywalker/src/offline_walk.cu
 */
#include "app.cuh"

__global__ void sample_kernel_escape(Walker *walker, float ratio) {
  Jobs_result<JobType::RW, uint> &result = walker->result;
  gpu_graph *graph = &walker->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  // TODO: add state for num runs
  size_t idx_i = TID;
  uint src = graph->adjsrc[idx_i];
  uint k = graph->adjk[idx_i];

  if (idx_i < result.size) {
    // printf("TID: %d, Src: %d (Degree: %d), Current: %d\n", (uint)idx_i, graph->adjsrc[TID], graph->getDegree(graph->adjsrc[TID]), graph->adjk[TID]);

    uint currentDegree = graph->getDegree(src);
    assert(currentDegree > 0);

    size_t numRuns = walker->num_runs / currentDegree;

    uint divider = (uint)1.0/ratio;
    if (divider > currentDegree)
      numRuns = divider; 

    uint sumPhiLocal = 0;
    
    for (size_t i = 0; i < numRuns; i++) {
      uint current = k;
      currentDegree = graph->getDegree(current);
      do {
        if(current == src) {
          sumPhiLocal += 1;
          break;
        }
        Vector_virtual<uint> alias;
        Vector_virtual<float> prob;
        alias.Construt(
            graph->alias_array + graph->xadj[current] - graph->local_vtx_offset,
            currentDegree);
        prob.Construt(
            graph->prob_array + graph->xadj[current] - graph->local_vtx_offset,
            currentDegree);
        alias.Init(currentDegree);
        prob.Init(currentDegree);
        const uint target_size = 1;
        if (target_size < currentDegree) {
          //   int itr = 0;
          // for (size_t i = 0; i < target_size; i++) {
          int col = (int)floor(hiprand_uniform(&state) * currentDegree);
          float p = hiprand_uniform(&state);
          uint candidate;
          if (p < prob[col])
            candidate = col;
          else
            candidate = alias[col];
            current = graph->getOutNode(current, candidate);
          // }
        } else if (currentDegree == 0) {
          result.alive[idx_i] = 0;
          break;
        } else {
          current = graph->getOutNode(current, 0);
        }
      } while (result.alive[idx_i] != 0 && current != graph->maxD);
    } 
  }
}

__global__ void sample_kernel_static_buffer(Walker *walker) {
  Jobs_result<JobType::RW, uint> &result = walker->result;
  gpu_graph *graph = &walker->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);
  __shared__ matrixBuffer<BLOCK_SIZE, 31, uint> buffer;
  buffer.Init();

  size_t idx_i = TID;
  if (idx_i < result.size) {
    result.length[idx_i] = result.hop_num - 1;
    uint src_id;
    // bool alive = true;
    coalesced_group warp = coalesced_threads();
    for (uint current_itr = 0; current_itr < result.hop_num - 1;
         current_itr++) {
      // coalesced_group active = coalesced_threads();
      if (result.alive[idx_i] != 0) {
        Vector_virtual<uint> alias;
        Vector_virtual<float> prob;
        src_id = current_itr == 0 ? result.GetData(current_itr, idx_i) : src_id;
        uint src_degree = graph->getDegree((uint)src_id);
        alias.Construt(
            graph->alias_array + graph->xadj[src_id] - graph->local_vtx_offset,
            src_degree);
        prob.Construt(
            graph->prob_array + graph->xadj[src_id] - graph->local_vtx_offset,
            src_degree);
        alias.Init(src_degree);
        prob.Init(src_degree);
        const uint target_size = 1;

        if (target_size < src_degree) {
          int col = (int)floor(hiprand_uniform(&state) * src_degree);
          float p = hiprand_uniform(&state);
          uint candidate;
          if (p < prob[col])
            candidate = col;
          else
            candidate = alias[col];
          uint next_src = graph->getOutNode(src_id, candidate);
          // if (idx_i == 1) printf("%u adding1 %u \n", idx_i, next_src);
          buffer.Set(next_src);
          src_id = next_src;
        } else if (src_degree == 0) {
          result.alive[idx_i] = 0;
          result.length[idx_i] = current_itr;
          // buffer.Finish();
          // return;
        } else {
          uint next_src = graph->getOutNode(src_id, 0);
          buffer.Set(next_src);
          src_id = next_src;
          // if (idx_i == 1) printf("%u adding %u \n", idx_i, next_src);
        }
      }
      warp.sync();
      buffer.CheckFlush(result.data + result.hop_num * idx_i, current_itr,
                        warp);
    }
    warp.sync();
    buffer.Flush(result.data + result.hop_num * idx_i, 0, warp);
  }
}
// 48 kb , 404 per sampler
__global__ void sample_kernel_static(Walker *walker) {
  Jobs_result<JobType::RW, uint> &result = walker->result;
  gpu_graph *graph = &walker->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  size_t idx_i = TID;
  if (idx_i < result.size) {
    result.length[idx_i] = result.hop_num - 1;
    for (uint current_itr = 0; current_itr < result.hop_num - 1;
         current_itr++) {
      if (result.alive[idx_i] != 0) {
        Vector_virtual<uint> alias;
        Vector_virtual<float> prob;
        uint src_id = result.GetData(current_itr, idx_i);
        uint src_degree = graph->getDegree((uint)src_id);
        alias.Construt(
            graph->alias_array + graph->xadj[src_id] - graph->local_vtx_offset,
            src_degree);
        prob.Construt(
            graph->prob_array + graph->xadj[src_id] - graph->local_vtx_offset,
            src_degree);
        alias.Init(src_degree);
        prob.Init(src_degree);
        const uint target_size = 1;
        if (target_size < src_degree) {
          //   int itr = 0;
          // for (size_t i = 0; i < target_size; i++) {
          int col = (int)floor(hiprand_uniform(&state) * src_degree);
          float p = hiprand_uniform(&state);
          uint candidate;
          if (p < prob[col])
            candidate = col;
          else
            candidate = alias[col];
          *result.GetDataPtr(current_itr + 1, idx_i) =
              graph->getOutNode(src_id, candidate);
          // }
        } else if (src_degree == 0) {
          result.alive[idx_i] = 0;
          result.length[idx_i] = current_itr;
          break;
        } else {
          *result.GetDataPtr(current_itr + 1, idx_i) =
              graph->getOutNode(src_id, 0);
        }
      }
    }
  }
}

__global__ void sample_kernel(Walker *walker) {
  Jobs_result<JobType::RW, uint> &result = walker->result;
  gpu_graph *graph = &walker->ggraph;
  hiprandState state;
  hiprand_init(TID, 0, 0, &state);

  for (size_t idx_i = TID; idx_i < result.size;
       idx_i += gridDim.x * blockDim.x) {
    result.length[idx_i] = result.hop_num - 1;
    for (uint current_itr = 0; current_itr < result.hop_num - 1;
         current_itr++) {
      if (result.alive[idx_i] != 0) {
        Vector_virtual<uint> alias;
        Vector_virtual<float> prob;
        uint src_id = result.GetData(current_itr, idx_i);
        uint src_degree = graph->getDegree((uint)src_id);
        alias.Construt(
            graph->alias_array + graph->xadj[src_id] - graph->local_vtx_offset,
            src_degree);
        prob.Construt(
            graph->prob_array + graph->xadj[src_id] - graph->local_vtx_offset,
            src_degree);
        alias.Init(src_degree);
        prob.Init(src_degree);
        const uint target_size = 1;
        if (target_size < src_degree) {
          //   int itr = 0;
          // for (size_t i = 0; i < target_size; i++) {
          int col = (int)floor(hiprand_uniform(&state) * src_degree);
          float p = hiprand_uniform(&state);
          uint candidate;
          if (p < prob[col])
            candidate = col;
          else
            candidate = alias[col];
          *result.GetDataPtr(current_itr + 1, idx_i) =
              graph->getOutNode(src_id, candidate);
          printf("Thread: %d edge: %d -> %d\n",TID, src_id, graph->getOutNode(src_id, candidate));
          // }
        } else if (src_degree == 0) {
          result.alive[idx_i] = 0;
          result.length[idx_i] = current_itr;
          break;
        } else {
          *result.GetDataPtr(current_itr + 1, idx_i) =
              graph->getOutNode(src_id, 0);
        }
      }
    }
  }
}

static __global__ void print_result(Walker *walker) {
  walker->result.PrintResult();
}

float OfflineWalk(Walker &walker) {
  LOG("%s\n", __FUNCTION__);
  int device;
  hipDeviceProp_t prop;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int n_sm = prop.multiProcessorCount;

  Walker *sampler_ptr;
  // printf("OfflineWalk xadj:\n");
  // for(size_t i = 0; i < 200; i++) {
  //   printf("%d, ", walker.ggraph.xadj[i]);
  // }
  // printf("\n\n");
  // printf("OfflineWalk adjsrc:\n");
  // for(size_t i = 0; i < 200; i++) {
  //   printf("%d, ", walker.ggraph.adjsrc[i]);
  // }
  // printf("\n\n");
  // printf("OfflineWalk adjk:\n");
  // for(size_t i = 0; i < 200; i++) {
  //   printf("%d, ", walker.ggraph.adjk[i]);
  // }
  // printf("\n\n");
  MyCudaMalloc(&sampler_ptr, sizeof(Walker));
  CUDA_RT_CALL(
      hipMemcpy(sampler_ptr, &walker, sizeof(Walker), hipMemcpyHostToDevice));
  double start_time, total_time;
  // init_kernel_ptr<<<1, 32, 0, 0>>>(sampler_ptr,true);
  BindResultKernel<<<1, 32, 0, 0>>>(sampler_ptr);
  // allocate global buffer
  int block_num = n_sm * FLAGS_m;
  CUDA_RT_CALL(hipDeviceSynchronize());
  CUDA_RT_CALL(hipPeekAtLastError());
  start_time = wtime();
#ifdef check
  sample_kernel<<<1, BLOCK_SIZE, 0, 0>>>(sampler_ptr);
#else
  if (FLAGS_static) {
    if (FLAGS_buffer)
      // sample_kernel_static_buffer<<<1, 32, 0, 0>>>(sampler_ptr);
      sample_kernel_static_buffer<<<walker.num_seed / BLOCK_SIZE + 1,
                                    BLOCK_SIZE, 0, 0>>>(sampler_ptr);
    else
      sample_kernel_static<<<walker.num_seed / BLOCK_SIZE + 1, BLOCK_SIZE, 0,
                             0>>>(sampler_ptr);
  } else if (FLAGS_escape) {
    LOG("Escape\n", __FUNCTION__);
    sample_kernel_escape<<<walker.num_seed / BLOCK_SIZE + 1, BLOCK_SIZE, 0,
                             0>>>(sampler_ptr, FLAGS_ratio);
  } else
    sample_kernel<<<block_num, BLOCK_SIZE, 0, 0>>>(sampler_ptr);
#endif
  CUDA_RT_CALL(hipDeviceSynchronize());
  // CUDA_RT_CALL(hipPeekAtLastError());
  total_time = wtime() - start_time;
#pragma omp barrier
  LOG("Device %d sampling time:\t%.6f ratio:\t %.2f MSEPS\n",
      omp_get_thread_num(), total_time,
      static_cast<float>(walker.result.GetSampledNumber() / total_time /
                         1000000));
  walker.sampled_edges = walker.result.GetSampledNumber();
  LOG("sampled_edges %d\n", walker.sampled_edges);
  if (FLAGS_printresult) print_result<<<1, 32, 0, 0>>>(sampler_ptr);
  CUDA_RT_CALL(hipDeviceSynchronize());
  return total_time;
}
